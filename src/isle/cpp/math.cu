#include "hip/hip_runtime.h"
#include "math.cuh"

namespace isle{

    CDMatrix mult_CDMatrix_wrapper(const CDMatrix &a, const CDMatrix &b, const std::size_t dim) {
        hipblasHandle_t handle;
        CHECK_CUBLAS_ERR(hipblasCreate(&handle));

        const hipDoubleComplex alpha = make_hipDoubleComplex(1,0), beta = make_hipDoubleComplex(0,0);
        const int N = static_cast<int>(dim);

        hipDoubleComplex * A;
        hipDoubleComplex * B;
        hipDoubleComplex * C;

        CHECK_CU_ERR(hipMalloc(&A, dim*dim*sizeof(hipDoubleComplex)));
        CHECK_CU_ERR(hipMalloc(&B, dim*dim*sizeof(hipDoubleComplex)));
        CHECK_CU_ERR(hipMalloc(&C, dim*dim*sizeof(hipDoubleComplex)));

        CHECK_CU_ERR(hipMemcpy(A, cast_cmpl(a.data()), dim*dim*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        CHECK_CU_ERR(hipMemcpy(B, cast_cmpl(b.data()), dim*dim*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

        // 2nd and 3rd arguments: HIPBLAS_OP_T means transposition which is needed for compatibility of blaze and cublas
        // 4-6,9,11,14-th arguments: matrix dimensions (all equal because matrices are square)
        // calculates C = a * A*B + b * C, with a=1 and b=0 in our case
        CHECK_CUBLAS_ERR(cublasZgemm3m(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha, A, N, B, N, &beta, C, N));

        CDMatrix res(a.rows(),a.columns());
        CHECK_CU_ERR(hipMemcpy(res.data(), cast_cmpl(C), dim*dim*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
        hipFree(A); hipFree(B); hipFree(C);

        CHECK_CUBLAS_ERR(hipblasDestroy(handle));

        blaze::transpose(res);
        return res;
    }

    void lu_CDMatrix_wrapper(CDMatrix &a, std::unique_ptr<int[]> &ipiv, const std::size_t dim) {
        hipsolverHandle_t handle;
        CHECK_CUSOLVER_ERR(hipsolverDnCreate(&handle));

        const int N = static_cast<int>(dim);
	int * d_ipiv;

        hipDoubleComplex * A;
        hipDoubleComplex * Workspace;

        CHECK_CU_ERR(hipMalloc(&A, dim*dim*sizeof(hipDoubleComplex)));
        CHECK_CU_ERR(hipMalloc(&d_ipiv, dim*sizeof(int)));

        CHECK_CU_ERR(hipMemcpy(A, cast_cmpl(a.data()), dim*dim*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

	int Lwork;
        CHECK_CUSOLVER_ERR(hipsolverDnZgetrf_bufferSize(handle, N, N, A, N, &Lwork));
        CHECK_CU_ERR(hipMalloc(&Workspace, static_cast<std::size_t>(Lwork)*sizeof(hipDoubleComplex)));

        int * devInfo;
        CHECK_CU_ERR(hipMallocManaged(&devInfo, sizeof(int)));
        // 2-3,5-th arguments: matrix dimensions (all equal because matrices are square)
        // calculates LU-decomposition of A inplace: ipiv * A = L * U 
        CHECK_CUSOLVER_ERR(hipsolverDnZgetrf(handle, N, N, A, N, Workspace, d_ipiv, devInfo));
        CHECK_CU_ERR(hipStreamSynchronize(0));
        assert(*devInfo == 0);

        CHECK_CU_ERR(hipMemcpy(a.data(), cast_cmpl(A), dim*dim*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
        CHECK_CU_ERR(hipMemcpy(ipiv.get(), d_ipiv, dim*sizeof(int), hipMemcpyDeviceToHost));
        hipFree(A); hipFree(Workspace); hipFree(d_ipiv); hipFree(devInfo);

        CHECK_CUSOLVER_ERR(hipsolverDnDestroy(handle));
    }

    void inv_CDMatrix_wrapper(const CDMatrix &a, CDMatrix &b, const std::unique_ptr<int[]> &ipiv, const std::size_t dim, const bool transpose) {
        hipsolverHandle_t handle;
        CHECK_CUSOLVER_ERR(hipsolverDnCreate(&handle));

        const int N = static_cast<int>(dim);
	int * d_ipiv;

        hipDoubleComplex * A;
        hipDoubleComplex * B;

        CHECK_CU_ERR(hipMalloc(&A, dim*dim*sizeof(hipDoubleComplex)));
        CHECK_CU_ERR(hipMalloc(&B, dim*dim*sizeof(hipDoubleComplex)));
        CHECK_CU_ERR(hipMalloc(&d_ipiv, dim*sizeof(int)));

	blaze::transpose(b);
        CHECK_CU_ERR(hipMemcpy(A, cast_cmpl(a.data()), dim*dim*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        CHECK_CU_ERR(hipMemcpy(B, cast_cmpl(b.data()), dim*dim*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        CHECK_CU_ERR(hipMemcpy(d_ipiv, ipiv.get(), dim*sizeof(int), hipMemcpyHostToDevice));

        hipblasOperation_t trans = transpose? HIPBLAS_OP_N : HIPBLAS_OP_T;
        int * devInfo;
        CHECK_CU_ERR(hipMallocManaged(&devInfo, sizeof(int)));
        // 2nd argument: HIPBLAS_OP_T(N) means (no) transposition of A, but one additional is needed for transfer from blaze to cublas
        // 3-4,6,9-th arguments: matrix dimensions (all equal because matrices are square)
        // solves A * x = B for x, where A has been LU-decomposed before
        CHECK_CUSOLVER_ERR(hipsolverDnZgetrs(handle, trans, N, N, A, N, d_ipiv, B, N, devInfo));
        CHECK_CU_ERR(hipStreamSynchronize(0));
        assert(*devInfo == 0);

        CHECK_CU_ERR(hipMemcpy(b.data(), cast_cmpl(B), dim*dim*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
        hipFree(A); hipFree(B); hipFree(d_ipiv); hipFree(devInfo);

	blaze::transpose(b);

        CHECK_CUSOLVER_ERR(hipsolverDnDestroy(handle));
    }

    __global__ void ilogdet_kernel(hipDoubleComplex * d_matrix, hipDoubleComplex * out){
        // adapted from https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

        extern __shared__ double sdata[];

        auto id = threadIdx.x;
        auto i = blockIdx.x*blockDim.x + threadIdx.x;

        sdata[id] = d_matrix[i];

        __synchthreads();

        for(unsigned int s = blockDim.x/2; s > 0; s>>=2){
            if (id < s){
                sdata[id] += log(sdata[id + s]);
            }
            __synchthreads();
        }

        if(id == 0){
            *out = sdata[0];
        }
    }

    std::complex<double> ilogdet_wrapper(std::complex<double> * matrix, std::size_t NX, bool & negDetP){
        hipblasHandle_t handle;
        CHECK_CUBLAS_ERR(hipblasCreate(&handle));

        int * ipiv;
        int * info;
        hipDoubleComplex * d_matrix;
        hipDoubleComplex * d_res;
        std::complex<double> res;
        CHECK_CU_ERR(hipMallocManaged(&ipiv, NX*sizeof(int)));
        CHECK_CU_ERR(hipMallocManaged(&d_res, sizeof(hipDoubleComplex)))
        CHECK_CU_ERR(hipMallocManaged(&info, sizeof(int)));
        CHECK_CU_ERR(hipMallocManaged(&d_matrix,NX*NX*sizeof(hipDoubleComplex)));

        CHECK_CU_ERR(hipMemcpy(d_matrix,matrix,NX*NX*sizeof(hipDoubleComplex),hipMemcpyHostToDevice));


        CHECK_CUBLAS_ERR(
            hipblasZgetrfBatched(
                handle,
                NX,
                d_matrix,
                1,
                ipiv,
                info,
                1
            )
        );

        auto num_blocks = ceildiv((int) NX,1024);

        ilogdet_kernel<<<dim3(num_blocks,1,1),dim3(1024,1,1)>>>(d_matrix,res);

        negDetP = false;  // if true det(P) == -1, else det(P) == +1
        for(int i = 0; i < NX; ++i){
            if (ipiv[i]-1 != i) {
                negDetP = !negDetP;
            }
        }

        auto res = cast_cmpl(*d_res);

        CHECK_CU_ERR(hipFree(ipiv));
        CHECK_CU_ERR(hipFree(d_matrix));

        CHECK_CUBLAS_ERR(hipblasDestroy(handle));

        return res;
    }
} // namespace isle
