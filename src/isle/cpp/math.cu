#include "math.cuh"

namespace isle{

    CDMatrix mult_CDMatrix_wrapper(const CDMatrix &a, const CDMatrix &b, const std::size_t dim) {
        hipblasHandle_t handle;
        CHECK_CUBLAS_ERR(hipblasCreate(&handle));

        const hipDoubleComplex alpha = make_hipDoubleComplex(1,0), beta = make_hipDoubleComplex(0,0);
        const int N = static_cast<int>(dim);

        hipDoubleComplex * A;
        hipDoubleComplex * B;
        hipDoubleComplex * C;

        CHECK_CU_ERR(hipMalloc(&A, dim*dim*sizeof(hipDoubleComplex)));
        CHECK_CU_ERR(hipMalloc(&B, dim*dim*sizeof(hipDoubleComplex)));
        CHECK_CU_ERR(hipMalloc(&C, dim*dim*sizeof(hipDoubleComplex)));

        CHECK_CU_ERR(hipMemcpy(A, cast_cmpl(a.data()), dim*dim*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        CHECK_CU_ERR(hipMemcpy(B, cast_cmpl(b.data()), dim*dim*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

        // 2nd and 3rd arguments: HIPBLAS_OP_T means transposition which is needed for compatibility of blaze and cublas
        // 4-6,9,11,14-th arguments: matrix dimensions (all equal because matrices are square)
        // calculates C = a * A*B + b * C, with a=1 and b=0 in our case
        CHECK_CUBLAS_ERR(cublasZgemm3m(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha, A, N, B, N, &beta, C, N));

        CDMatrix res(a.rows(),a.columns()); 
        CHECK_CU_ERR(hipMemcpy(res.data(), cast_cmpl(C), dim*dim*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
        hipFree(A); hipFree(B); hipFree(C);

        CHECK_CUBLAS_ERR(hipblasDestroy(handle));

        return res;
    }

    void lu_CDMatrix_wrapper(CDMatrix &a, std::unique_ptr<int[]> &ipiv, const std::size_t dim) {
        hipsolverHandle_t handle;
        CHECK_CUSOLVER_ERR(hipsolverDnCreate(&handle));

        const int N = static_cast<int>(dim);
	int * d_ipiv;

        hipDoubleComplex * A;
        hipDoubleComplex * Workspace;

        CHECK_CU_ERR(hipMalloc(&A, dim*dim*sizeof(hipDoubleComplex)));
        CHECK_CU_ERR(hipMalloc(&d_ipiv, dim*sizeof(int)));

        CHECK_CU_ERR(hipMemcpy(A, cast_cmpl(a.data()), dim*dim*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

	int Lwork;
        CHECK_CUSOLVER_ERR(hipsolverDnZgetrf_bufferSize(handle, N, N, A, N, &Lwork));
        CHECK_CU_ERR(hipMalloc(&Workspace, static_cast<std::size_t>(Lwork)*sizeof(hipDoubleComplex)));

        int * devInfo;
        CHECK_CU_ERR(hipMallocManaged(&devInfo, sizeof(int)));
        // 2nd argument: HIPBLAS_OP_T(N) means (no) transposition, but one additional is needed for transfer from blaze to cublas
        // 4-6,9,11,14-th arguments: matrix dimensions (all equal because matrices are square)
        // calculates C = a * A*B + b * C, with a=1 and b=0 in our case
        CHECK_CUSOLVER_ERR(hipsolverDnZgetrf(handle, N, N, A, N, Workspace, d_ipiv, devInfo));
        hipDeviceSynchronize();
        assert(*devInfo == 0);

        CHECK_CU_ERR(hipMemcpy(a.data(), cast_cmpl(A), dim*dim*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
        CHECK_CU_ERR(hipMemcpy(ipiv.get(), d_ipiv, dim*sizeof(int), hipMemcpyDeviceToHost));
        hipFree(A); hipFree(Workspace); hipFree(d_ipiv); hipFree(devInfo);

        CHECK_CUSOLVER_ERR(hipsolverDnDestroy(handle));
    }

    void inv_CDMatrix_wrapper(const CDMatrix &a, CDMatrix &b, const std::unique_ptr<int[]> &ipiv, const std::size_t dim, const bool transpose) {
        hipsolverHandle_t handle;
        CHECK_CUSOLVER_ERR(hipsolverDnCreate(&handle));

        const int N = static_cast<int>(dim);
	int * d_ipiv;

        hipDoubleComplex * A;
        hipDoubleComplex * B;

        CHECK_CU_ERR(hipMalloc(&A, dim*dim*sizeof(hipDoubleComplex)));
        CHECK_CU_ERR(hipMalloc(&B, dim*dim*sizeof(hipDoubleComplex)));
        CHECK_CU_ERR(hipMalloc(&d_ipiv, dim*sizeof(int)));

	blaze::transpose(b);
        CHECK_CU_ERR(hipMemcpy(A, cast_cmpl(a.data()), dim*dim*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        CHECK_CU_ERR(hipMemcpy(B, cast_cmpl(b.data()), dim*dim*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        CHECK_CU_ERR(hipMemcpy(d_ipiv, ipiv.get(), dim*sizeof(int), hipMemcpyHostToDevice));

        hipblasOperation_t trans = transpose? HIPBLAS_OP_N : HIPBLAS_OP_T;
        int * devInfo;
        CHECK_CU_ERR(hipMallocManaged(&devInfo, sizeof(int)));
        // 2nd argument: HIPBLAS_OP_T(N) means (no) transposition, but one additional is needed for transfer from blaze to cublas
        // 4-6,9,11,14-th arguments: matrix dimensions (all equal because matrices are square)
        // calculates C = a * A*B + b * C, with a=1 and b=0 in our case
        CHECK_CUSOLVER_ERR(hipsolverDnZgetrs(handle, trans, N, N, A, N, d_ipiv, B, N, devInfo));
        hipDeviceSynchronize();
        assert(*devInfo == 0);

        CHECK_CU_ERR(hipMemcpy(b.data(), cast_cmpl(B), dim*dim*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
        hipFree(A); hipFree(B); hipFree(d_ipiv); hipFree(devInfo);

	blaze::transpose(b);

        CHECK_CUSOLVER_ERR(hipsolverDnDestroy(handle));
    }
} // namespace isle
