#include "math.cuh"

namespace isle{

    CDMatrix mult_CDMatrix_wrapper(const CDMatrix &a, const CDMatrix &b, const std::size_t dim) {
        hipblasHandle_t handle;
        CHECK_CUBLAS_ERR(hipblasCreate(&handle));

        const hipDoubleComplex alpha = make_hipDoubleComplex(1,0), beta = make_hipDoubleComplex(0,0);
        const int N = static_cast<int>(dim);

        hipDoubleComplex * A;
        hipDoubleComplex * B;
        hipDoubleComplex * C;

        CHECK_CU_ERR(hipMalloc(&A, dim*dim*sizeof(hipDoubleComplex)));
        CHECK_CU_ERR(hipMalloc(&B, dim*dim*sizeof(hipDoubleComplex)));
        CHECK_CU_ERR(hipMalloc(&C, dim*dim*sizeof(hipDoubleComplex)));

        CHECK_CU_ERR(hipMemcpy(A, cast_cmpl(a.data()), dim*dim*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        CHECK_CU_ERR(hipMemcpy(B, cast_cmpl(b.data()), dim*dim*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

        // 2nd and 3rd arguments: HIPBLAS_OP_T means transposition which is needed for compatibility of blaze and cublas
        // 4-6,9,11,14-th arguments: matrix dimensions (all equal because matrices are square)
        // calculates C = a * A*B + b * C, with a=1 and b=0 in our case
        CHECK_CUBLAS_ERR(cublasZgemm3m(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha, A, N, B, N, &beta, C, N));

        CDMatrix res = a; // WARNING: This is very dirty and has to be removed later.
        CHECK_CU_ERR(hipMemcpy(res.data(), cast_cmpl(C), dim*dim*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
        hipFree(A); hipFree(B); hipFree(C);

        CHECK_CUBLAS_ERR(hipblasDestroy(handle));

    return res;
  }
} // namespace isle
